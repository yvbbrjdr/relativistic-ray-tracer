#include "hip/hip_runtime.h"
#include <stdio.h>
#include <hiprand/hiprand_kernel.h>
#include "setup.h"

#define MAX_NUM_LIGHT 20
#define MAX_NUM_BSDF 20

#define RUSSIAN_ROULETTE
#define SHADOW_RAY

#define INF_FLOAT 1e20
#define ESP_N 5e-3
#define EPS_K 1e-4

#define BLOCK_DIM 64
#define LEAF_NUMBER 4

__constant__  GPUCamera const_camera;
__constant__  GPUBSDF const_bsdfs[MAX_NUM_BSDF];
__constant__  GPULight const_lights[MAX_NUM_LIGHT];
__constant__  Parameters const_params;
__constant__  BVHParameters const_bvhparams;


__device__ void
generateRay(GPURay* ray, float x, float y)
{
    ray->depth = 0;
    ray->min_t = 0;
    ray->max_t = 1e10;
    float sp[3];
    float dir[3];
    initVector3D(-(x - 0.5) * const_camera.widthDivDist,
    -(y - 0.5) * const_camera.heightDivDist, 1, sp);
    negVector3D(sp, dir);
    MatrixMulVector3D(const_camera.c2w, sp, ray->o);
    addVector3D(const_camera.pos, ray->o);
    MatrixMulVector3D(const_camera.c2w, dir, ray->d);
    normalize3D(ray->d);
}

__device__ float3
tracePixel(int x, int y)
{
   float3 spec = make_float3(1.0, 0.0, 0.0);
   int w = const_params.screenW;
   int h = const_params.screenH;
   for (int i = 0; i < 5; i++)
   {
     float px = x / (float)w;
     float py = y / (float)h;
     GPURay ray;
     generateRay(&ray, px, py);
   }

   return spec;
}

__global__ void
traceScene()
{
    int index = blockDim.x * blockIdx.x + threadIdx.x;

    if (index >= const_params.screenW * const_params.screenH) {
        return;
    }

    hiprandState s;
    hiprand_init((unsigned int)index, 0, 0, &s);

    int x = index % const_params.screenW;
    int y = index / const_params.screenW;

    tracePixel(x, y);

    const_params.frameBuffer[3 * index] = 1.0;
    const_params.frameBuffer[3 * index + 1] = 0.5;
    const_params.frameBuffer[3 * index + 2] = 0.5;
}


__device__ float2 gridSampler(hiprandState *s) {
    float2 rt;
    rt.x = hiprand_uniform(s);
    rt.y = hiprand_uniform(s);
    return rt;
}

__device__ bool sphereTest(int primIndex, GPURay& ray, double& t1, double& t2) {
    float* primitive = const_params.positions + 9 * primIndex;
    float* o = primitive;
    float r = primitive[3];
    float r2 = r * r;

     float m[3];
    subVector3D(o, ray.o, m);
    double b = VectorDot3D(m, ray.d);
    double c = VectorDot3D(m, m) - r2;
    double delta = b * b - c;
    if (delta < 0) {
        return false;
    }

     t1 = b - sqrt(delta);
    t2 = b + sqrt(delta);

     if (t1 >= ray.max_t || t2 <= ray.min_t) {
        return false;
    }

     return true;
}

 __device__ bool sphereIntersect(int primIndex, GPURay& r) {
    double tmp;
    return sphereTest(primIndex, r, tmp, tmp);
}

 __device__ bool sphereIntersect(int primIndex, GPURay& r, GPUIntersection *isect) {
    double t1;
    double t2;
    bool res = sphereTest(primIndex, r, t1, t2);
    if (!res) {
        return false;
    }
    isect->bsdfIndex = const_params.bsdfIndexes[primIndex];
    isect->pIndex = primIndex;

     float* primitive = const_params.positions + 9 * primIndex;
    float* o = primitive;
    double t = t1;
    if (t1 <= r.min_t) {
        t = t2;
    }
    float n[3];
    float tmp[3];
    for (int i = 0; i < 3; ++i)
    {
        tmp[i] = r.d[i] * t;
    }
    addVector3D(r.o, tmp);
    subVector3D(tmp, o, n);
    normalize3D(n);
    readVector3D(n, isect->n);
    isect->t = t;
    r.max_t = t;

     return true;
}

__device__ bool triangleIntersect(int primIndex, GPURay& r, GPUIntersection *isect) {

     float* primitive = const_params.positions + 9 * primIndex;
    float* normals = const_params.normals + 9 * primIndex;

     float* v1 = primitive;
    float* v2 = primitive + 3;
    float* v3 = primitive + 6;

     float e1[3], e2[3], s[3];
    subVector3D(v2, v1, e1);
    subVector3D(v3, v1, e2);
    subVector3D(r.o, v1, s);

     float tmp[3];
    VectorCross3D(e1, r.d, tmp);
    double f = VectorDot3D(tmp, e2);
    if (f == 0) {
        return false;
    }

     VectorCross3D(s, r.d, tmp);
    double u = VectorDot3D(tmp, e2) / f;
    VectorCross3D(e1, r.d, tmp);
    double v = VectorDot3D(tmp, s) / f;
    VectorCross3D(e1, s, tmp);
    double t = - VectorDot3D(tmp, e2) / f;

     if (!(u >= 0 && v >= 0 && u+v <= 1 && t > r.min_t && t < r.max_t && t < isect->t)) {
        return false;
    }

     r.max_t = t;

     isect->bsdfIndex = const_params.bsdfIndexes[primIndex];
    isect->t = t;
    isect->pIndex = primIndex;

     float *n1 = normals;
    float *n2 = normals + 3;
    float *n3 = normals + 6;

     float n[3];
    for (int i = 0; i < 3; ++i)
    {
        n[i] = (1 - u - v) * n1[i] + u * n2[i] + v * n3[i];
    }
    if (VectorDot3D(r.d, n) > 0)
    {
        negVector3D(n, n);
    }
    readVector3D(n, isect->n);

     return true;
}

__global__ void
vectorAdd(float *A, float *B, float *C, int numElements)
{
    int i = blockDim.x * blockIdx.x + threadIdx.x;

    if (i < numElements)
    {
        gpuAdd(A + i, B + i, C + i);
        //C[i] = A[i] + B[i];
    }
}
