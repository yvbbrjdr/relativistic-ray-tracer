#include "hip/hip_runtime.h"
/**
 * Copyright 1993-2015 NVIDIA Corporation.  All rights reserved.
 *
 * Please refer to the NVIDIA end user license agreement (EULA) associated
 * with this source code for terms and conditions that govern your use of
 * this software. Any use, reproduction, disclosure, or distribution of
 * this software and related documentation outside the terms of the EULA
 * is strictly prohibited.
 *
 */

/**
 * Vector addition: C = A + B.
 *
 * This sample is a very basic sample that implements element by element
 * vector addition. It is the same as the sample illustrating Chapter 2
 * of the programming guide with some additions like error checking.
 */

#include <stdio.h>

// For the CUDA runtime routines (prefixed with "cuda_")
#include <hip/hip_runtime.h>

extern __global__ void VectorAdd(float *A, float *B, float *C, int numElements)
{
	int i = blockDim.x * blockIdx.x + threadIdx.x;

	if (i < numElements)
	{
		gpuAdd(A + i, B + i, C + i);
	}
}
