#include "hip/hip_runtime.h"
#include <stdio.h>
#include <hiprand/hiprand_kernel.h>
#include "setup.h"

#define MAX_NUM_LIGHT 20
#define MAX_NUM_BSDF 20

#define RUSSIAN_ROULETTE
#define SHADOW_RAY

#define INF_FLOAT 1e20
#define ESP_N 5e-3
#define EPS_K 1e-4

#define BLOCK_DIM 64
#define LEAF_NUMBER 4

__constant__  GPUCamera const_camera;
__constant__  GPUBSDF const_bsdfs[MAX_NUM_BSDF];
__constant__  GPULight const_lights[MAX_NUM_LIGHT];
__constant__  Parameters const_params;
__constant__  BVHParameters const_bvhparams;

__global__ void
tracePixel()
{
    int index = blockDim.x * blockIdx.x + threadIdx.x;

     if (index >= const_params.screenW * const_params.screenH) {
        return;
    }

    const_params.frameBuffer[3 * index] = 1.0;
    const_params.frameBuffer[3 * index + 1] = 0.5;
    const_params.frameBuffer[3 * index + 2] = 0.5;
    hiprandState s;
    hiprand_init((unsigned int)index, 0, 0, &s);
}

void CUDAPathTracer::startRayTracing()
{
    int blockDim = 256;
    int gridDim = (const_params.screenW * const_params.screenH + blockDim - 1) / blockDim;

     tracePixel<<<gridDim, blockDim>>>();
    hipDeviceSynchronize();
}

__device__ float2 gridSampler(hiprandState *s) {
    float2 rt;
    rt.x = hiprand_uniform(s);
    rt.y = hiprand_uniform(s);
    return rt;
}

__global__ void
vectorAdd(float *A, float *B, float *C, int numElements)
{
    int i = blockDim.x * blockIdx.x + threadIdx.x;

    if (i < numElements)
    {
        gpuAdd(A + i, B + i, C + i);
        //C[i] = A[i] + B[i];
    }
}
